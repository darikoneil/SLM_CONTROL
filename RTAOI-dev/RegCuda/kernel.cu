#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h> 
#include <> 
#include <> 
#include <builtin_types.h> 
#include <vector_functions.h> 
#include <float.h>

#define _SIZE_T_DEFINED 
#ifndef __HIPCC__ 
#define __HIPCC__ 
#endif 
#ifndef __cplusplus 
#define __cplusplus 
#endif

typedef __int32 int32_t;
typedef unsigned __int32 uint32_t;
// Texture reference

extern "C"
{
#include <hip/hip_complex.h>
#define MATRIX_SIZE  512
#define	TILE_SIZE 32
#define BLOCK_SIZE 32

	__global__ void MatrixMulKernel(hipComplex *A, hipComplex *B, hipComplex *C)
	{
		// inner product of A and B.conj()
		const int W = MATRIX_SIZE;

		// Block index
		const int bx = blockIdx.x;
		const int by = blockIdx.y;

		// Thread index
		const int tx = threadIdx.x;
		const int ty = threadIdx.y;

		// Index of the first sub-matrix 
		const int INDEX = W * BLOCK_SIZE * bx + BLOCK_SIZE *by + W*tx + ty;

		hipComplex conjB = make_hipComplex(hipCimagf(B[INDEX]), hipCrealf(B[INDEX]));
		hipComplex Csub = hipCmulf(A[INDEX], conjB);
		C[INDEX] = make_hipComplex(hipCrealf(Csub), hipCimagf(Csub));

	}

	__global__ void int2complex(hipFloatComplex *A, int32_t *B)
	{
		const int W = MATRIX_SIZE;

		// Block index
		const int bx = blockIdx.x;
		const int by = blockIdx.y;

		// Thread index
		const int tx = threadIdx.x;
		const int ty = threadIdx.y;

		// Index of the first sub-matrix 
		const int INDEX = W * BLOCK_SIZE * bx + BLOCK_SIZE *by + W*tx + ty;

		A[INDEX] = make_hipComplex(float(B[INDEX]), 0);

	}

	__global__ void float2complex(hipFloatComplex *A, float *B)
	{
		const int W = MATRIX_SIZE;

		// Block index
		const int bx = blockIdx.x;
		const int by = blockIdx.y;

		// Thread index
		const int tx = threadIdx.x;
		const int ty = threadIdx.y;

		// Index of the first sub-matrix 
		const int INDEX = W * BLOCK_SIZE * bx + BLOCK_SIZE *by + W*tx + ty;

		A[INDEX] = make_hipFloatComplex(B[INDEX], 0);

	}

	__global__ void abs_of_complex(hipComplex *A, float *B)
	{
		const int W = MATRIX_SIZE;

		// Block index
		const int bx = blockIdx.x;
		const int by = blockIdx.y;

		// Thread index
		const int tx = threadIdx.x;
		const int ty = threadIdx.y;

		// Index of the first sub-matrix 
		const int INDEX = W * BLOCK_SIZE * bx + BLOCK_SIZE*by + W*tx + ty;

		B[INDEX] = hipCabsf(A[INDEX]);

	}

	__global__ void sample_mean(short * matrix, int pixelsPerLine,
		int linesPerFrame, int samplesPerPixel, int flipEvenRows, int32_t* result)
	{
		int idx = threadIdx.x + blockDim.x*blockIdx.x;
		int result_idx = 0;
		int col = 0;
		int num_sample = 0;
		int this_value = 0;
		
		if (idx<pixelsPerLine*linesPerFrame*samplesPerPixel - samplesPerPixel+1){
			if ((idx - (idx / samplesPerPixel)*samplesPerPixel) == 0){
				result_idx = idx / samplesPerPixel;
				col = result_idx - (result_idx / pixelsPerLine)*pixelsPerLine;
				if ((result_idx / pixelsPerLine) - ((result_idx / pixelsPerLine) / 2) * 2 != flipEvenRows){
					result_idx = result_idx + pixelsPerLine - 2 * col - 1;
				}

				for (int i = 0; i < samplesPerPixel; i++){
					if (matrix[idx + i]>8192){
						this_value += matrix[idx + i] - 8192;
						num_sample += 1;
					}
				}

				if (num_sample>0){ result[result_idx] = this_value / num_sample; }
			}
		}
	}

	__global__ void sample_mean_debug(short * matrix, float* result)
	{
		int idx = threadIdx.x + blockDim.x*blockIdx.x;
		int result_idx = 0;
		int col = 0;
		int num_sample = 0;
		int this_value = 0;

		if (idx<512*512*3){
			if ((idx - (idx /3)*3) == 0){
				result_idx = idx / 3;
				col = result_idx - (result_idx / 512)*512;
				if ((result_idx / 512) - ((result_idx / 512) / 2) * 2 != 1){
					result_idx = result_idx + 512 - 2 * col - 1;
				}

				for (int i = 0; i < 3; i++){
					if (matrix[idx + i]>8192){
						this_value += matrix[idx + i] - 8192;
						num_sample += 1;
					}
				}
				result[result_idx] = 0;
				if (num_sample>0){ result[result_idx] = (float)this_value / (float)num_sample; }
				

			}
		}
	}
}